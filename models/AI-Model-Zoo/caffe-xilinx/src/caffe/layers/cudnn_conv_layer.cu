
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/cudnn_conv_layer.hpp"
#include "caffe/util/gpu_memory.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

__global__ void sync_conv_groups() {}

template<typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  int device;
  CUDA_CHECK(cudaGetDevice(&device));
  size_t& WORKSPACE_SIZE = workspace_size(device);
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();

    // Sometimes closer to zero we might have memory info diverged from reality
    // If try_reserve fails, it updates the info internally and we proceed with
    // Reshape one more time
    // Note: if WORKSPACE_SIZE is already allocated next line does nothing.
    if (!WORKSPACE.try_reserve(WORKSPACE_SIZE)) {
      use_algo_seeker_ = true;
      this->Reshape(bottom, top);
      WORKSPACE.reserve(WORKSPACE_SIZE);
    }

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(Caffe::cudnn_handle(),
          cudnn::dataType<Dtype>::one,
          bottom_descs_[i], bottom_data + bottom_offset_ * g,
          filter_desc_, weight + this->weight_offset_ * g,
          conv_descs_[i],
          fwd_algo_[i], WORKSPACE.data(), WORKSPACE.size(),
          cudnn::dataType<Dtype>::zero,
          top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            bias_desc_, bias_data + bias_offset_ * g,
            cudnn::dataType<Dtype>::one,
            top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
  }
  // Possibly use faster algorithms by allowing larger workspace.
  use_modest_workspace_ = false;
}

template<typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  int device;
  CUDA_CHECK(cudaGetDevice(&device));
  size_t& WORKSPACE_SIZE = workspace_size(device);
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();

    // Sometimes closer to zero we might have memory info diverged from reality
    // If try_reserve fails, it updates the info internally and we proceed with
    // Reshape one more time.
    // Note: if WORKSPACE_SIZE is already allocated next line does nothing.
    if (!WORKSPACE.try_reserve(WORKSPACE_SIZE)) {
      use_algo_seeker_ = true;
      this->Reshape(bottom, top);
      WORKSPACE.reserve(WORKSPACE_SIZE);
    }

    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            top_descs_[i], top_diff + top_offset_ * g,
            cudnn::dataType<Dtype>::one,
            bias_desc_, bias_diff + bias_offset_ * g));
      }
      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            top_descs_[i], top_diff + top_offset_ * g,
            conv_descs_[i],
            bwd_filter_algo_[i], WORKSPACE.data(), WORKSPACE.size(),
            cudnn::dataType<Dtype>::one,
            filter_desc_, weight_diff + this->weight_offset_ * g));
      }
      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(Caffe::cudnn_handle(),
            cudnn::dataType<Dtype>::one,
            filter_desc_, weight + this->weight_offset_ * g,
            top_descs_[i], top_diff + top_offset_ * g,
            conv_descs_[i],
            bwd_data_algo_[i], WORKSPACE.data(), WORKSPACE.size(),
            cudnn::dataType<Dtype>::zero,
            bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
