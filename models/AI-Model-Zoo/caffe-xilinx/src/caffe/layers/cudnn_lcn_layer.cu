
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_lcn_layer.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNLCNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  temp1_.reserve(tempDataSize_);
  temp2_.reserve(tempDataSize_);

  CUDNN_CHECK(cudnnDivisiveNormalizationForward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL,  // srcMeansData
        temp1_.data(), temp2_.data(),
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );

  temp1_.release();
  temp2_.release();
}

template <typename Dtype>
void CuDNNLCNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  temp1_.reserve(tempDataSize_);
  temp2_.reserve(tempDataSize_);

  CUDNN_CHECK(cudnnDivisiveNormalizationBackward(
        Caffe::cudnn_handle(), norm_desc_,
        CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL, top_diff,  // NULL - srcMeansData
        temp1_.data(), temp2_.data(),
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff,
        NULL) );

  temp1_.release();
  temp2_.release();
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLCNLayer);

}  // namespace caffe
#endif
