#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/gs_tiling_layer.hpp"

namespace caffe {

    template <typename Dtype>
    __global__ void tiling_kernel(const Dtype *x, int wi, int hi, int ci, int wo, int ho, int co, int batch, int stride, int forward, Dtype *out)
    {
        int size = batch*ci*hi*wi;
        int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
        if(i >= size) return;
        int iw = i % wi;
        i = i / wi;
        int ih = i % hi;
        i = i / hi;
        int ic = i % ci;
        i = i / ci;
        int ib = i % batch;
        int ow = iw / stride;
        int oh = ih / stride;
        int oc = ((ih % stride) * stride + (iw % stride)) * ci + ic;
        int oi = ((ib * co + oc) * ho + oh) * wo + ow;
        int ii = ((ib * ci + ic) * hi + ih) * wi + iw;


        if(forward)
        {
            out[oi] = x[ii];
        }         
        else
        {
            out[ii] = x[oi];
        }
    }


template <typename Dtype>
void GSTilingLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype *bottom_data = bottom[0]->gpu_data();
        int count = bottom[0]->count();
        Dtype *top_data = top[0]->mutable_gpu_data();
	int batch_num_ = bottom[0]->num();
        if(reverse_)
        {
	        tiling_kernel<Dtype>
	         <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(bottom_data, output_width_, output_height_,
	                  output_channels_, input_width_, input_height_, input_channels_, batch_num_, stride_, !reverse_, top_data);

        }else
	    {
	        tiling_kernel<Dtype>
	         <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(bottom_data, input_width_, input_height_,
	                  input_channels_, output_width_, output_height_, output_channels_, batch_num_, stride_, !reverse_, top_data);
	    }
}

template <typename Dtype>
void GSTilingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if(!propagate_down[0]){
            return;
        }
        int count = top[0]->count();
	int batch_num_ = bottom[0]->num();
        const Dtype *top_diff = top[0]->gpu_diff();
        Dtype *bottom_diff = bottom[0]->mutable_gpu_diff();
        if(reverse_)
        {
	        tiling_kernel<Dtype>
	         <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(top_diff, output_width_, output_height_,
	                  output_channels_, input_width_, input_height_, input_channels_, batch_num_, stride_, reverse_, bottom_diff);

        }else
	    {
	        tiling_kernel<Dtype>
	         <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(top_diff, input_width_, input_height_,
	                  input_channels_, output_width_, output_height_, output_channels_, batch_num_, stride_, reverse_, bottom_diff);
	    }
}

INSTANTIATE_LAYER_GPU_FUNCS(GSTilingLayer);

}  // namespace caffe
