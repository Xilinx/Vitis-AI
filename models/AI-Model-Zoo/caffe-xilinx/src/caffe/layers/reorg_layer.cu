#include "hip/hip_runtime.h"
#include "caffe/layers/reorg_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {
    template <typename Dtype>
    __global__ void reorg_kernel(const Dtype *x, int w, int h, int c, int batch, int stride, int forward, Dtype *out)
    {
        int size = batch*c*h*w;
        int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
        if(i >= size) return;
        int in_index = i;
        int in_w = i%w;
        i = i/w;
        int in_h = i%h;
        i = i/h;
        int in_c = i%c;
        i = i/c;
        int b = i%batch;

        int out_c = c/(stride*stride);

        int c2 = in_c % out_c;
        int offset = in_c / out_c;
        int w2 = in_w*stride + offset % stride;
        int h2 = in_h*stride + offset / stride;
        int out_index = w2 + w*stride*(h2 + h*stride*(c2 + out_c*b));

        if(forward)
        {
            out[out_index] = x[in_index];
        }         
        else
        {
            out[in_index] = x[out_index];
        }
    }

    template<typename Dtype>
    void ReorgLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype> *> &bottom,
                                        const vector<Blob<Dtype> *> &top) {
        const Dtype *bottom_data = bottom[0]->gpu_data();
        int count = bottom[0]->count();
        Dtype *top_data = top[0]->mutable_gpu_data();
        reorg_kernel<Dtype>
         <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(bottom_data, width_, height_,
                  channels_, batch_num_, stride_, reverse_, top_data);
    }

    template<typename Dtype>
    void ReorgLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype> *> &top, const vector<bool> &propagate_down,
                                         const vector<Blob<Dtype> *> &bottom) {
        if(!propagate_down[0]){
            return;
        }
        int count = bottom[0]->count();
        const Dtype *top_diff = top[0]->gpu_diff();
        Dtype *bottom_diff = bottom[0]->mutable_gpu_diff();
        reorg_kernel<Dtype>
         <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(top_diff, width_, height_,
                  channels_, batch_num_, stride_, !reverse_, bottom_diff);
    }

INSTANTIATE_LAYER_GPU_FUNCS(ReorgLayer);

}  // namespace caffe
