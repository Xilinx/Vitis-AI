#include "hip/hip_runtime.h"


/*
* Copyright 2019 Xilinx Inc.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <math.h>
#include <algorithm>
#include <hip/hip_math_constants.h>
#include "../include/nndct_fix_kernels.cuh"
#include "../include/nndct_cu_utils.h"
#include "../include/nndct_cuda_math.h"

template<typename Dtype>
__global__ static void _set(const int N, 
                            Dtype* data, 
                            Dtype val){
  NNDCT_KERNEL_LOOP(index, N){
    data[index] = val;
  }
}

template<typename Dtype>
__global__ static void _scale_inplace(const int N, 
                                      Dtype* data, 
                                      Dtype scale){
  NNDCT_KERNEL_LOOP(index, N){
    data[index] *= scale;
  }
}

template<typename Dtype>
__global__ static void _scale(const int N,
                              const Dtype* src,
                              Dtype* dst,
                              Dtype scale){
  NNDCT_KERNEL_LOOP(index, N){
    dst[index] = scale * src[index];
  }
}

template<typename Dtype>
__global__ static void _sub(const int N, 
                            const Dtype* src, 
                            Dtype* dst){
  NNDCT_KERNEL_LOOP(index, N){
    dst[index] = src[index] - dst[index];
  }
}

template<typename Dtype>
__global__ static void _pow(const int N,
                            Dtype* data,
                            Dtype power){
  NNDCT_KERNEL_LOOP(index, N){
    data[index] = pow(data[index], power);
  }
}

//from kaldi, reduction without device handle
enum EnumTransformReduce {
  SUMAB, SUM, MAX, MIN, LINFNORM, L2NORM, L1NORM, L0NORM, LPNORM
};

template<EnumTransformReduce TransReduceType, typename Dtype>
struct TransReduceOp {
  __forceinline__
  __device__ Dtype InitValue() const {
    return Dtype(0);
  }
  __forceinline__
  __device__ Dtype Transform(const Dtype& x) const {
    return Dtype(0);
  }
  __forceinline__
  __device__ Dtype Reduce(const Dtype& a, const Dtype& b) const {
    return Dtype(0);
  }
  __forceinline__
  __device__ Dtype PostReduce(const Dtype& x, const Dtype& output) const {
    return Dtype(0);
  }
};

template<typename Dtype>
struct TransReduceOp<SUM, Dtype> {
  __forceinline__
  __device__ Dtype InitValue() const {
    return Dtype(0);
  }
  __forceinline__
  __device__ Dtype Transform(const Dtype& x) const {
    return x;
  }
  __forceinline__
  __device__ Dtype Reduce(const Dtype& a, const Dtype& b) const {
    return a + b;
  }
  __forceinline__
  __device__ Dtype PostReduce(const Dtype& x, const Dtype& output) const {
    return x;
  }
};

template<typename Dtype>
struct TransReduceOp<MAX, Dtype> {
  __forceinline__
  __device__ Dtype InitValue() const {
    return sizeof(Dtype) == sizeof(float) ? -HIP_INF_F : -HIP_INF;
  }
  __forceinline__
  __device__ Dtype Transform(const Dtype& x) const {
    return x;
  }
  __forceinline__
  __device__ Dtype Reduce(const Dtype& a, const Dtype& b) const {
    return fmax(a, b);
  }
  __forceinline__
  __device__ Dtype PostReduce(const Dtype& x, const Dtype& output) const {
    return x;
  }
};

template<typename Dtype>
struct TransReduceOp<MIN, Dtype> {
  __forceinline__
  __device__ Dtype InitValue() const {
    return sizeof(Dtype) == sizeof(float) ? HIP_INF_F : HIP_INF;
  }
  __forceinline__
  __device__ Dtype Transform(const Dtype& x) const {
    return x;
  }
  __forceinline__
  __device__ Dtype Reduce(const Dtype& a, const Dtype& b) const {
    return min(a, b);
  }
  __forceinline__
  __device__ Dtype PostReduce(const Dtype& x, const Dtype& output) const {
    return x;
  }
};

template<EnumTransformReduce TransReduceType, typename Dtype>
__global__
static void _vec_transform_reduce(const int dim,const Dtype* src, Dtype* dst,
  const TransReduceOp<TransReduceType, Dtype> op) {
  
  __shared__ Dtype sdata[CU1DBLOCK];
  Dtype tdata = op.InitValue();

  const int tid = threadIdx.x;
  const int vec_len = dim;
  const int grid_stride = gridDim.x * blockDim.x;
  int i = (blockIdx.x * blockDim.x + tid);
  
  // Grid reduce. Loop over the whole vector v.
  for (; i < vec_len; i += grid_stride) {
    tdata = op.Reduce(tdata, op.Transform(src[i]));
  }
  
  sdata[tid] = tdata;
  __syncthreads();

  // Tree reduce
# pragma unroll
  for (int shift = CU1DBLOCK / 2; shift > warpSize; shift >>= 1) {
    if (tid < shift) {
      sdata[tid] = op.Reduce(sdata[tid], sdata[tid + shift]);
    }
    __syncthreads();
  }

  // Reduce last warp. Threads implicitly synchronized within a warp.
  if (tid < warpSize) {
    for (int shift = warpSize; shift > 0; shift >>= 1) {
      sdata[tid] = op.Reduce(sdata[tid], sdata[tid + shift]);
    }
  }
  
  // Output to vector dst.
  if (tid == 0)
    dst[blockIdx.x] = op.PostReduce(sdata[0], dst[blockIdx.x]);
}

template<EnumTransformReduce TransReduceType, typename Dtype>
__global__
static void _vec_transform_reduce_inplace(const int dim,Dtype* data,
  const TransReduceOp<TransReduceType, Dtype> op) {
  
  __shared__ Dtype sdata[CU1DBLOCK];
  Dtype tdata = op.InitValue();

  const int tid = threadIdx.x;
  const int vec_len = dim;
  const int grid_stride = gridDim.x * blockDim.x;
  int i = (blockIdx.x * blockDim.x + tid);
  
  // Grid reduce. Loop over the whole vector v.
  for (; i < vec_len; i += grid_stride) {
    tdata = op.Reduce(tdata, op.Transform(data[i]));
    data[i]=0;
  }
  
  sdata[tid] = tdata;
  __syncthreads();

  // Tree reduce
# pragma unroll
  for (int shift = CU1DBLOCK / 2; shift > warpSize; shift >>= 1) {
    if (tid < shift) {
      sdata[tid] = op.Reduce(sdata[tid], sdata[tid + shift]);
    }
    __syncthreads();
  }

  // Reduce last warp. Threads implicitly synchronized within a warp.
  if (tid < warpSize) {
    for (int shift = warpSize; shift > 0; shift >>= 1) {
      sdata[tid] = op.Reduce(sdata[tid], sdata[tid + shift]);
    }
  }
  
  // Output to vector dst.
  if (tid == 0)
    data[blockIdx.x] = op.PostReduce(sdata[0], data[blockIdx.x]);
}

template<EnumTransformReduce TransReduceType, typename Dtype>
__global__ static void _single_reduce(const int dim, Dtype* dst,
  const TransReduceOp<TransReduceType, Dtype> op){
  for(int i = 1; i < dim; i++){
    dst[0] = op.Reduce(dst[0], dst[i]);
    dst[i] = 0;
  }
}

template<typename Dtype>
void cuda_set(const int N, Dtype* data, Dtype val){
  _set<<<NNDCT_GET_BLOCKS(N), NNDCT_CUDA_NUM_THREADS>>>(
    N, data, val);
}
template 
void cuda_set<float>(const int N, float* data, float val);
template 
void cuda_set<double>(const int N, double* data, double val);
  

template<typename Dtype>
void cuda_scale_inplace(const int N, Dtype* data, Dtype scale){
  _scale_inplace<<<NNDCT_GET_BLOCKS(N), NNDCT_CUDA_NUM_THREADS>>>(
    N, data, scale);
}
template
void cuda_scale_inplace<float>(const int N, float* data, float scale);
template
void cuda_scale_inplace<double>(const int N, double* data, double scale);


template<typename Dtype>
void cuda_scale(const int N, const Dtype* src, Dtype* dst, Dtype scale){
  _scale<<<NNDCT_GET_BLOCKS(N), NNDCT_CUDA_NUM_THREADS>>>(
    N, src, dst, scale);
}
template
void cuda_scale<float>(const int N, const float* src, float* dst, float scale);
template
void cuda_scale<double>(const int N, const double* src, double* dst, double scale);


template<typename Dtype>
void cuda_pow(const int N, Dtype* data, Dtype pow){
  _pow<<<NNDCT_GET_BLOCKS(N), NNDCT_CUDA_NUM_THREADS>>>(
    N, data, pow);
}
template
void cuda_pow<float>(const int N, float* data, float pow);
template
void cuda_pow<double>(const int N, double* data, double pow);


template<typename Dtype>
void cuda_max(const int N, const Dtype* src, Dtype* dst){
  int dimGrid=NNDCT_GET_BLOCKS1D(N);
  _vec_transform_reduce<<<dimGrid, CU1DBLOCK>>>(
    N, src, dst, TransReduceOp<MAX, Dtype>());

  _single_reduce<<<1, 1>>>(
    dimGrid, dst, TransReduceOp<MAX, Dtype>());
}
template
void cuda_max<float>(const int N, const float* src, float* dst);
template
void cuda_max<double>(const int N, const double* src, double* dst);


template<typename Dtype>
void cuda_min(const int N, const Dtype* src, Dtype* dst){
  int dimGrid=NNDCT_GET_BLOCKS1D(N);
  _vec_transform_reduce<<<dimGrid, CU1DBLOCK>>>(
    N, src, dst, TransReduceOp<MIN, Dtype>());

  _single_reduce<<<1, 1>>>(
    dimGrid, dst, TransReduceOp<MIN, Dtype>());
}
template
void cuda_min<float>(const int N, const float* src, float* dst);
template
void cuda_min<double>(const int N, const double* src, double* dst);


template<typename Dtype>
void cuda_sum(const int N, const Dtype* src, Dtype* dst){
  int dimGrid=NNDCT_GET_BLOCKS1D(N);
  _vec_transform_reduce<<<dimGrid,CU1DBLOCK>>>(
    N, src, dst, TransReduceOp<SUM, Dtype>());

  _single_reduce<<<1, 1>>>(
    dimGrid, dst, TransReduceOp<SUM, Dtype>());
}
template
void cuda_sum<float>(const int N, const float* src, float* dst);
template
void cuda_sum<double>(const int N, const double* src, double* dst);


template<typename Dtype>
void cuda_sum_inplace(const int N, Dtype* data){
  int dimGrid = NNDCT_GET_BLOCKS1D(N);
  _vec_transform_reduce_inplace<<<dimGrid, CU1DBLOCK>>>(
    N, data, TransReduceOp<SUM, Dtype>());

  _single_reduce<<<1, 1>>>(
    dimGrid, data, TransReduceOp<SUM, Dtype>());
}
template
void cuda_sum_inplace<float>(const int N, float* data);
template
void cuda_sum_inplace<double>(const int N, double* data);


template<typename Dtype>
void cuda_sub(const int N, const Dtype* src, Dtype* dst){
  _sub<<<NNDCT_GET_BLOCKS(N), NNDCT_CUDA_NUM_THREADS>>>(
      N, src, dst);
}
template
void cuda_sub<float>(const int N, const float* src, float* dst);
template
void cuda_sub<double>(const int N, const double* src, double* dst);

